#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "conv.h"

__global__ void apply_kernel_cuda(double *d_grayscale, double *d_output, int width, int height, double *d_kernel) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        double sum = 0.0;
        for (int ky = 0; ky < KERNEL_SIZE; ky++) {
            for (int kx = 0; kx < KERNEL_SIZE; kx++) {
                int px = x + kx - KERNEL_SIZE / 2;
                int py = y + ky - KERNEL_SIZE / 2;
                if (px >= 0 && px < width && py >= 0 && py < height) {
                    sum += d_grayscale[py * width + px] * d_kernel[ky * KERNEL_SIZE + kx];
                }
            }
        }
        d_output[y * width + x] = sum;
    }
}

extern "C" void apply_kernel_cuda_wrapper(double *grayscale, double *output, int width, int height, double **kernel, int block_size) {
    double *d_grayscale, *d_output, *d_kernel;
    size_t size = width * height * sizeof(double);
    size_t kernel_size = KERNEL_SIZE * KERNEL_SIZE * sizeof(double);

    // Allocate device memory
    hipMalloc((void **)&d_grayscale, size);
    hipMalloc((void **)&d_output, size);
    hipMalloc((void **)&d_kernel, kernel_size);

    // Copy data from host to device
    hipMemcpy(d_grayscale, grayscale, size, hipMemcpyHostToDevice);

    // Flatten the kernel for copying to device
    double *flattened_kernel = (double *)malloc(kernel_size);
    for (int i = 0; i < KERNEL_SIZE; i++) {
        for (int j = 0; j < KERNEL_SIZE; j++) {
            flattened_kernel[i * KERNEL_SIZE + j] = kernel[i][j];
        }
    }
    hipMemcpy(d_kernel, flattened_kernel, kernel_size, hipMemcpyHostToDevice);
    free(flattened_kernel);

    // Define block and grid sizes
    dim3 threadsPerBlock(block_size, block_size);
    dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x, (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Launch the kernel
    apply_kernel_cuda<<<numBlocks, threadsPerBlock>>>(d_grayscale, d_output, width, height, d_kernel);

    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch kernel: %s\n", hipGetErrorString(err));
        hipFree(d_grayscale);
        hipFree(d_output);
        hipFree(d_kernel);
        return;
    }

    // Copy data from device to host
    hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_grayscale);
    hipFree(d_output);
    hipFree(d_kernel);

    // Reset the device to clear the cache
    hipDeviceReset();
}